#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

#define uint unsigned int
#define uchar unsigned char

#define rows 288
#define cols 384

uchar *gpu_data;
uchar *out;

dim3 threads(16,16);
dim3 blocks(18,24);

__global__ void box_kernel_x(uchar *input,uchar *output,int r)
{
	uint x=(blockIdx.x*blockDim.x)+threadIdx.x;
	uint y=(blockIdx.y*blockDim.y)+threadIdx.y;
	uint offset=y+x*blockDim.y*gridDim.y;
	if((x>=r)&&(x<rows-1-r)&&(y>=r)&&(y<cols-1-r))
	{
		int sum=0;
		for(int i=x-r;i<x+r+1;i++)
		{
			sum+=input[i*cols+y];
		}
		output[offset]=sum/(r<<1+1);
	}
	else
		output[offset]=0;
}

__global__ void box_kernel_y(uchar *input,uchar *output,int r)
{
	uint x=(blockIdx.x*blockDim.x)+threadIdx.x;
	uint y=(blockIdx.y*blockDim.y)+threadIdx.y;
	uint offset=y+x*blockDim.y*gridDim.y;
	if((x>=r)&&(x<rows-1-r)&&(y>=r)&&(y<cols-1-r))
	{
		int sum=0;
		for(int j=y-r;j<y+r+1;j++)
		{
			sum+=input[x*cols+j];
		}
		output[offset]=sum/(r<<1+1);
	}
	else
		output[offset]=0;
}

void boxfilter(uchar *input,uchar *output,int r,dim3 block,dim3 thread)
{
	uchar *temp1;
	hipMalloc((void **)&temp1,sizeof(uchar)*rows*cols);
	box_kernel_x<<<block,thread>>>(input,temp1,r);
	box_kernel_y<<<block,thread>>>(temp1,output,r);
}

int main()
{
	//不使用纹理内存
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	Mat img=imread("im2.ppm",0);
	imshow("原图像",img);
	hipMalloc((void **)&gpu_data,sizeof(uchar)*rows*cols);
	hipMalloc((void **)&out,sizeof(uchar)*rows*cols);
	hipMemcpy(gpu_data,img.data,sizeof(uchar)*rows*cols,hipMemcpyHostToDevice);
	hipEventRecord(start,0);

	boxfilter(gpu_data,out,7,blocks,threads);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float time;
	hipEventElapsedTime(&time,start,stop);
	hipMemcpy(img.data,out,sizeof(uchar)*rows*cols,hipMemcpyDeviceToHost);
	imshow("不使用纹理内存盒式滤波后的图像",img);
	cout<<"不使用纹理内存所用时间:"<<time<<"ms"<<endl;
	hipFree(gpu_data);
	hipFree(out);

	waitKey(0);
	return 0;
}

